#include "hip/hip_runtime.h"
#include "wm_common.h"
#include "wall_model_options.h"
#include "wall_model_worker.h"
#include "wm_lib_typedef.h"
#include "hybrid_computing.h"
#include <stdio.h>
#include <iostream>
#include "setup.h"
#include <stdlib.h>
#include "mpi.h"
#include <atomic>
#include <unistd.h>
#include <iostream>
#include "GlobalMemHandler.h"
namespace wall_model_module
{
#define CU_ERROR_CHECK_WMK(something) CudaKillIfError_WMK(something, __FILE__, __LINE__, &node_info)
	void CudaKillIfError_WMK(hipError_t _cu_error, const char* file, const int line, NodeCommHandler* node_info)
	{
		if (_cu_error != hipSuccess)
		{
			if (node_info->is_node_root_process)
			{
				std::cout << ">>>>>>>>>>> Fatal CUDA runtime error on node ";
				std::cout << node_info->node_name << " (" << std::hex << node_info->processor_node_id << std::dec << "), process " << node_info->global_proc_info.process_id << ":" << std::endl;
				std::cout << ">>>>>>>>>>> File: " << file << ", line " << line << std::endl;
				std::cout << ">>>>>>>>>>> Error: " << hipGetErrorString(_cu_error) << std::endl;
				abort();
			}
		}
	}
	__device__ int compute_global_index(void)
	{
		//for now
		return threadIdx.x;
	}

	__global__ void K_pass_global(HybridComputeInstance<__dev_real*> K_globaldata)
	{
		HYBRID::globaldata = K_globaldata;
	}

	//I think this needs some restructuring!! One definition rule.
	__global__ void K_compute_solution_MODEL_ALGEBRAIC()
	{
		HYBRID::compute_solution_MODEL_ALGEBRAIC<__hybrid>(compute_global_index());
	}

	__global__ void K_init_solution_MODEL_ALGEBRAIC()
	{
		HYBRID::init_solution_MODEL_ALGEBRAIC<__hybrid>(compute_global_index());
	}



	template <const int NUM_EQS> __global__ void K_compute_solution_MODEL_ODE()
	{
		HYBRID::compute_solution_MODEL_ODE<__hybrid, NUM_EQS>(compute_global_index());
	}
	template __global__ void K_compute_solution_MODEL_ODE<2>();
	template __global__ void K_compute_solution_MODEL_ODE<3>();


	template <const int NUM_EQS> __global__ void K_init_solution_MODEL_ODE()
	{
		HYBRID::init_solution_MODEL_ODE<__hybrid, NUM_EQS>(compute_global_index(), WMODE_INIT_TAU_RATIO);
	}
	template __global__ void K_init_solution_MODEL_ODE<2>();
	template __global__ void K_init_solution_MODEL_ODE<3>();

	void launch_kernel_MODEL_ALGEBRAIC(HybridComputeInstance<__dev_real*> globaldata)
	{
		K_pass_global<<<1,1>>>(globaldata);
		if (MemHandler.node_total_gpu_allocation > 0)
		{
			MPI_Barrier(node_info.gpualloc_comm);
			MemHandler.data_transfer<double>(hipMemcpyHostToDevice);
			MPI_Barrier(node_info.gpualloc_comm);
			if (gpu_solution_init_required && node_info.is_node_root_process)
			{
				K_init_solution_MODEL_ALGEBRAIC<<<1, MemHandler.node_total_gpu_allocation>>>();
				CU_ERROR_CHECK_WMK(hipPeekAtLastError());
				CU_ERROR_CHECK_WMK(hipDeviceSynchronize());
			}
			gpu_solution_init_required = false;
			if (node_info.is_node_root_process)
			{
				K_compute_solution_MODEL_ALGEBRAIC<<<1, MemHandler.node_total_gpu_allocation>>>();
				CU_ERROR_CHECK_WMK(hipPeekAtLastError());
			}
		}
	}

	void launch_kernel_MODEL_ODE(HybridComputeInstance<__dev_real*> globaldata)
	{
		K_pass_global<<<1,1>>>(globaldata);
		if (MemHandler.node_total_gpu_allocation > 0)
		{
			MPI_Barrier(node_info.gpualloc_comm);
			MemHandler.data_transfer<double>(hipMemcpyHostToDevice);
			MPI_Barrier(node_info.gpualloc_comm);
			if (user_settings.include_energy_equation)
			{
				if (gpu_solution_init_required && node_info.is_node_root_process)
				{
					K_init_solution_MODEL_ODE<2><<<1, MemHandler.node_total_gpu_allocation>>>();
					CU_ERROR_CHECK_WMK(hipPeekAtLastError());
					CU_ERROR_CHECK_WMK(hipDeviceSynchronize());
				}
				gpu_solution_init_required = false;
				if (node_info.is_node_root_process)
				{
					K_compute_solution_MODEL_ODE<2><<<1, MemHandler.node_total_gpu_allocation>>>();
					CU_ERROR_CHECK_WMK(hipPeekAtLastError());
				}
			}
			else
			{
				if (gpu_solution_init_required && node_info.is_node_root_process)
				{
					K_init_solution_MODEL_ODE<3><<<1, MemHandler.node_total_gpu_allocation>>>();
					CU_ERROR_CHECK_WMK(hipPeekAtLastError());
					CU_ERROR_CHECK_WMK(hipDeviceSynchronize());
				}
				gpu_solution_init_required = false;
				if (node_info.is_node_root_process)
				{
					K_compute_solution_MODEL_ODE<3><<<1, MemHandler.node_total_gpu_allocation>>>();
					CU_ERROR_CHECK_WMK(hipPeekAtLastError());
				}
			}
		}
	}

	void solve_gpu_allocation(void)
	{
		if (MemHandler.gpu_allocation > 0)
		{
			WmInstanceData instance;
			instance.is_on_gpu = true;
			instance.num_wall_points = MemHandler.gpu_allocation;
			instance.ode_convection_pg = false;

			HybridComputeInstance<__dev_real*> globaldata;
	        globaldata.instance = instance;
	        globaldata.buffer = gpu_buffers;
	        globaldata.settings = user_settings;

			switch(user_settings.model_selection)
			{
				default:
				case MODEL_ALGEBRAIC:
				{
					launch_kernel_MODEL_ALGEBRAIC(globaldata);
					break;
				}
				case MODEL_ODE_PRESSURE_GRADIENT_CONVECTION:
				{
					instance.ode_convection_pg = true;
					launch_kernel_MODEL_ODE(globaldata);
					break;
				}
				case MODEL_ODE_NO_PRESSURE_GRADIENT_NO_CONVECTION:
				{
					launch_kernel_MODEL_ODE(globaldata);
					break;
				}
			}
		}
	}

	void offload_gpu_solution(void)
	{
		//MPI_Barrier(node_info.gpualloc_comm); // <-- this took ~7 hours to track down.
		if (MemHandler.node_total_gpu_allocation > 0)
		{
			CU_ERROR_CHECK_WMK(hipDeviceSynchronize()); // THIS MIGHT BE AN ISSUE!!!!!!
			MPI_Barrier(node_info.gpualloc_comm);
			MemHandler.data_transfer<double>(hipMemcpyDeviceToHost);
			MPI_Barrier(node_info.gpualloc_comm);
		}
	}
}
